#include "hip/hip_runtime.h"
//
// Created by Dillon Yao on 4/26/17.
//


#include <glm/gtc/type_ptr.hpp>
#include "ParticleManager.h"
#include "../util.h"
#include "../display/Application.h"
#include "../cuda/CudaPBDSolver.cuh"

#define M 512

ParticleManager::ParticleManager() {}

void ParticleManager::init() {

    particle_radius = PARTICLE_RADIUS;
    shade_mode = SHADE_PARTICLE;
    skybox_id = parent->skybox.textureID;

    int nx = 30;
    int ny = 30;
    int nz = 30;

    float d = particle_radius * 2;
    for (int x = 0; x < nx; ++x) {
        for (int y = 1 / d; y < 1 / d + ny; ++y) {
            for (int z = 0; z < nz; ++z) {
                Particle par;
                par.p = vec3((x + 0.5 - nx * 0.5) * d, y * d, (z + 0.5 - nz * 0.5) * d);
                par.pred_p = glm::vec3();
                par.v = glm::vec3();
                par.m = PARTICLE_MASS;
                par.num_neighbors = 0;
                par.lambda = 0;
                par.collided = false;
                particles.push_back(par);
                initial_positions.push_back(par.p);
            }
        }
    }

    Plane ground; //BOTTOM
    ground.point = glm::vec3(0, 0, 0);
    ground.normal = glm::vec3(0, 1, 0);

    Plane side0; // RIGHT
    side0.point = glm::vec3(5, 0, 0);
    side0.normal = glm::vec3(1, 0, 0);

    Plane side1; //BACK
    side1.point = glm::vec3(0, 0, 5);
    side1.normal = glm::vec3(0, 0, 1);

    Plane side2; //LEFT
    side2.point = glm::vec3(-10, 0, 0);
    side2.normal = glm::vec3(1, 0, 0);

    Plane side3; // BACK
    side3.point = glm::vec3(0, 0, -5);
    side3.normal = glm::vec3(0, 0, 1);

    Plane side4; // TOP
    side4.point = glm::vec3(0, 10, 0);
    side4.normal = glm::vec3(0, 1, 0);
    planes.push_back(ground);
    planes.push_back(side0);
    planes.push_back(side1);
    planes.push_back(side2);
    planes.push_back(side3);
    planes.push_back(side4);

    particle_shader.load("src/shaders/particle.vert", "src/shaders/particle.frag");
    velocity_shader.load("src/shaders/particle_velocity.vert", "src/shaders/particle_velocity.frag");
    water_shader.load("src/shaders/water.vert", "src/shaders/water.frag");

    Light &light = this->parent->light;

    particle_shader.use();
    GLint lightPosLoc = glGetUniformLocation(particle_shader.program, "light_pos");
    glUniform3f(lightPosLoc, light.pos.x, light.pos.y, light.pos.z);
    GLint lightColorLoc = glGetUniformLocation(particle_shader.program, "light_color");
    glUniform3f(lightColorLoc, light.color.r, light.color.g, light.color.b);

    velocity_shader.use();
    lightPosLoc = glGetUniformLocation(particle_shader.program, "light_pos");
    glUniform3f(lightPosLoc, light.pos.x, light.pos.y, light.pos.z);
    lightColorLoc = glGetUniformLocation(particle_shader.program, "light_color");
    glUniform3f(lightColorLoc, light.color.r, light.color.g, light.color.b);

    water_shader.use();
    lightPosLoc = glGetUniformLocation(water_shader.program, "light_pos");
    glUniform3f(lightPosLoc, light.pos.x, light.pos.y, light.pos.z);
    lightColorLoc = glGetUniformLocation(water_shader.program, "light_color");
    glUniform3f(lightColorLoc, light.color.r, light.color.g, light.color.b);

    particle_mesh = generate_sphere_mesh(PARTICLE_RADIUS * 0.9f, 10, 10);
    glGenBuffers(1, &instanceVBO);
    glBindVertexArray(particle_mesh.VAO);

    glBindBuffer(GL_ARRAY_BUFFER, instanceVBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(Particle) * particles.size(), NULL, GL_STREAM_DRAW);

    glEnableVertexAttribArray(3);
    glVertexAttribPointer(3, 3, GL_FLOAT, GL_FALSE, sizeof(Particle), (GLvoid *) offsetof(Particle, p));
    glVertexAttribDivisor(3, 1);

    glEnableVertexAttribArray(4);
    glVertexAttribPointer(4, 3, GL_FLOAT, GL_FALSE, sizeof(Particle), (GLvoid *) offsetof(Particle, v));
    glVertexAttribDivisor(4, 1);

    glBindBuffer(GL_ARRAY_BUFFER, 0);
    glBindVertexArray(0);

    triangles = &parent->test_model.triangles;

    hipMalloc((void **) &d_particles, particles.size() * sizeof(Particle));
    hipMalloc((void **) &d_triangles, triangles->size() * sizeof(Triangle));
    hipMalloc((void **) &d_planes, planes.size() * sizeof(Plane));

    hipMemcpy(d_planes, &planes[0], planes.size() * sizeof(Plane), hipMemcpyHostToDevice);
    hipMemcpy(d_triangles, &triangles->data()[0], triangles->size() * sizeof(Triangle), hipMemcpyHostToDevice);
}

void ParticleManager::render(Camera &c, mat4 &projection, mat4 &view) {
    glBindBuffer(GL_ARRAY_BUFFER, instanceVBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(Particle) * particles.size(), NULL, GL_STREAM_DRAW);
    glBufferSubData(GL_ARRAY_BUFFER, offsetof(Particle, p), particles.size() * sizeof(Particle), &particles[0]);

    bind_shader(projection, view, c.pos);
    glBindVertexArray(particle_mesh.VAO);
    glBindTexture(GL_TEXTURE_CUBE_MAP, skybox_id);
    glDrawElementsInstanced(GL_TRIANGLES, particle_mesh.indices.size(), GL_UNSIGNED_INT, 0, particles.size());
    glBindVertexArray(0);
}

void ParticleManager::step(float dt) {
    hipMemcpy(d_particles, &particles[0], particles.size() * sizeof(Particle), hipMemcpyHostToDevice);
    seed_position<<<(particles.size() + M-1) / M, M>>>(d_particles, particles.size());
    hipDeviceSynchronize();    
    hipMemcpy(&particles[0], d_particles, particles.size() * sizeof(Particle), hipMemcpyDeviceToHost);

    spacial_map.clear();
    for (int i = 0; i < particles.size(); ++i) {
        int hash = hash_bin(bin(particles[i]));
        if (spacial_map.find(hash) == spacial_map.end()) {
            spacial_map[hash] = new std::vector<int>();
        }
        spacial_map[hash]->push_back(i);
    }

    for (Particle &p : particles) {
        for (int n : this->neighborhood(p)) {
            if (p.num_neighbors < 50)
                p.neighborhood[p.num_neighbors++] = n;
        }
    }

    hipMemcpy(d_particles, &particles[0], particles.size() * sizeof(Particle), hipMemcpyHostToDevice);
    run_solver<<<(particles.size() + M-1) / M, M>>>(d_particles, particles.size(), d_triangles, triangles->size(), d_planes, planes.size());
    hipDeviceSynchronize();
    hipMemcpy(&particles[0], d_particles, particles.size() * sizeof(Particle), hipMemcpyDeviceToHost);
}

glm::ivec3 ParticleManager::bin(Particle& p) {
    double d = KERNEL_RADIUS; // TODO Change d to private var bin_size
    int bin_x = std::floor(p.pred_p.x / d);
    int bin_y = std::floor(p.pred_p.y / d);
    int bin_z = std::floor(p.pred_p.z / d);
    return glm::ivec3(bin_x, bin_y, bin_z);
}

int ParticleManager::hash_bin(glm::ivec3 pos) {
    return (pos.x * 0x9e3779b9 + pos.y) * 1610612741 + pos.z;
}

std::vector<int> ParticleManager::neighborhood(Particle& p) {
    std::vector<int> neighbors;
    glm::ivec3 original_bin = this->bin(p);
    for (int i = -1; i <= 1; ++i) {
        for (int j = -1; j <= 1; ++j) {
            for (int k = -1; k <= 1; ++k) {
                glm::ivec3 bin = original_bin + glm::ivec3(i, j, k);
                int hash = hash_bin(bin);
                if (spacial_map.find(hash) == spacial_map.end()) {
                    continue;
                }
                for (int n : *spacial_map[hash]) {
                    Particle &neighbor = particles[n];
                    if (&neighbor != &p && glm::length(neighbor.pred_p - p.pred_p) <= KERNEL_RADIUS) {
                        neighbors.push_back(n);
                    }
                }
            }
        }
    }
    return neighbors;
}

void ParticleManager::next_mode() {
    shade_mode = (shade_mode + 1) % SHADERS_TOTAL;
}

void ParticleManager::bind_shader(glm::mat4 &projection, glm::mat4 &view, glm::vec3 &view_pos) {
    switch(shade_mode) {
        case SHADE_PARTICLE: {
            particle_shader.use();
            glUniform3f(glGetUniformLocation(particle_shader.program, "view_pos"), view_pos.x, view_pos.y, view_pos.z);
            glUniformMatrix4fv(glGetUniformLocation(particle_shader.program, "projection"), 1, GL_FALSE, value_ptr(projection));
            glUniformMatrix4fv(glGetUniformLocation(particle_shader.program, "view"), 1, GL_FALSE, value_ptr(view));
            break;
        }

        case SHADE_VELOCITY: {
            velocity_shader.use();
            glUniform3f(glGetUniformLocation(velocity_shader.program, "view_pos"), view_pos.x, view_pos.y, view_pos.z);
            glUniformMatrix4fv(glGetUniformLocation(velocity_shader.program, "projection"), 1, GL_FALSE, value_ptr(projection));
            glUniformMatrix4fv(glGetUniformLocation(velocity_shader.program, "view"), 1, GL_FALSE, value_ptr(view));
            break;
        }

        case SHADE_WATER: {
            water_shader.use();
            glUniform3f(glGetUniformLocation(water_shader.program, "view_pos"), view_pos.x, view_pos.y, view_pos.z);
            glUniformMatrix4fv(glGetUniformLocation(water_shader.program, "projection"), 1, GL_FALSE, value_ptr(projection));
            glUniformMatrix4fv(glGetUniformLocation(water_shader.program, "view"), 1, GL_FALSE, value_ptr(view));
            glActiveTexture(GL_TEXTURE0);
            glUniform1i(glGetUniformLocation(water_shader.program, "skybox"), 0);
            break;
        }
    }
}

void ParticleManager::set_parent(Application *app) {
    this->parent = app;
}

void ParticleManager::reset() {
    auto par = particles.begin();
    auto pos = initial_positions.begin();
    for (;par != particles.end();) {
        par->p = *pos;
        par->pred_p = glm::dvec3();
        par->v = glm::dvec3();
        par->num_neighbors = 0;
        par->lambda = 0;
        ++par;
        ++pos;
    }
}
