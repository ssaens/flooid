#include "hip/hip_runtime.h"


#define DELTA_T 0.004f
#define PI 3.14159265359f
#define SOLVER_ITERS 4
#define KERNEL_RADIUS 0.1001f
#define EPS_T 600.f
#define REST_DENSITY 6378.f
#define PRESSURE_STRENGTH 0.000001f
#define PRESSURE_POW 4.f;
#include "CudaPBDSolver.cuh"

const float VISCOSITY = 0.01;

const float SURFACE_OFFSET = 0.000001;

using namespace glm;

__device__ inline float poly6(glm::vec3 r_ij, float h);

__device__ inline float spiky(glm::vec3 r_ij, float h);

__device__ inline glm::vec3 spiky_grad(glm::vec3 r_ij, float h);

__device__ float rho_i(Particle *p_i, Particle *particles);

__device__ float lambda_i(Particle *p_i, Particle *particles);

__device__ float C_i(Particle *p_i, Particle *particles);

__device__ glm::vec3 grad_k_Ci(Particle *p_k, Particle *p_i, Particle *particles);

__device__ glm::vec3 delta_p(Particle *p_i, Particle *particles);

__device__ glm::vec3 vorticity(Particle *p_i, Particle *particles);

__device__ glm::vec3 f_vorticity(Particle *p_i, Particle *particles);

__device__ glm::vec3 XSPH_vel(Particle *p_i, Particle *particles);

__device__ void triangle_collide(Triangle &t, Particle &p);

__device__ void plane_collide(Plane &p, Particle &par);

__global__ void seed_position(Particle *particles, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= n) {
        return;
    }
    Particle &p = particles[index];
    p.f = glm::vec3(0, -9.8f, 0) * p.m;
    p.v = p.v + p.f * (1.f / p.m) * DELTA_T;
    p.pred_p = p.p + p.v * DELTA_T;
}

__global__ void run_solver(Particle *particles, int n, Triangle *triangles, int num_triangles, Plane *planes, int num_planes) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= n) {
        return;
    }
    Particle &p = particles[index];

    for (int i = 0; i < SOLVER_ITERS; ++i) {
        __syncthreads();
        p.lambda = lambda_i(&p, particles);
        __syncthreads();

        p.dp = delta_p(&p, particles);
        p.pred_p += p.dp;
        for (int i = 0; i < num_triangles; ++i) {
            triangle_collide(triangles[i], p);
        }
        for (int i = 0; i < num_planes; ++i) {
            plane_collide(planes[i], p);
        }
    }

    if (!p.collided)
        p.v = (1.f / DELTA_T) * (p.pred_p - p.p);
    __syncthreads();

    p.f += f_vorticity(&p, particles);
    p.v = XSPH_vel(&p, particles);
    p.collided = false;
    p.p = p.pred_p;
    p.num_neighbors = 0;
}

__device__ float poly6(glm::vec3 r_ij, float h) {
    float r = length(r_ij);
    return 0 <= r && r <= h ? 315 / (64 * PI * pow(h, 9)) * pow(h * h - r * r, 3) : 0;
}

__device__ float spiky(glm::vec3 r_ij, float h) {
    float r = length(r_ij);
    return 0 <= r && r <= h ? 15 / (PI * pow(h, 6)) * pow(h - r, 3) : 0;
}

__device__ glm::vec3 spiky_grad(glm::vec3 r_ij, float h) {
    float r = length(r_ij);
    if (0 < r && r <= h) {
        glm::vec3 d = normalize(r_ij);
        float coeff = 45 / (PI * pow(h, 6)) * pow(h - r, 2);
        return coeff * d;
    } else {
        return glm::vec3();
    }
}

__device__ float rho_i(Particle *p_i, Particle *particles) {
    float rho = 0;
    for (int i = 0; i < p_i->num_neighbors; ++i) {
        int n = p_i->neighborhood[i];
        Particle *p_j = &particles[n];
        rho += p_j->m * poly6(p_i->pred_p - p_j->pred_p, KERNEL_RADIUS);
    }
    return rho;
}

__device__ float lambda_i(Particle *p_i, Particle *particles) {
    float lambda = 0;
    for (int i = 0; i < p_i->num_neighbors; ++i) {
        int n = p_i->neighborhood[i];
        Particle *p_k = &particles[n];
        glm::vec3 grad_k_ci = grad_k_Ci(p_k, p_i, particles);
        lambda += pow(length(grad_k_ci), 2);
    }
    lambda = -C_i(p_i, particles) * (1 / (lambda + EPS_T));
    return lambda;
}

__device__ float C_i(Particle *p_i, Particle *particles) {
    return rho_i(p_i, particles) / REST_DENSITY - 1;
}

__device__ glm::vec3 grad_k_Ci(Particle *p_k, Particle *p_i, Particle *particles) {
    if (p_k == p_i) {
        glm::vec3 grad_spiky;
        for (int i = 0; i < p_i->num_neighbors; ++i) {
            int n = p_i->neighborhood[i];
            Particle *p_j = &particles[n];
            grad_spiky += spiky_grad(p_i->pred_p - p_j->pred_p, KERNEL_RADIUS);
        }
        return (1 / REST_DENSITY) * grad_spiky;
    } else {
        return -(1 / REST_DENSITY) * spiky_grad(p_i->pred_p - p_k->pred_p, KERNEL_RADIUS);
    }
}

__device__ glm::vec3 delta_p(Particle *p_i, Particle *particles) {
    glm::vec3 delta_p;
    float s_corr;
    for (int i = 0; i < p_i->num_neighbors; ++i) {
        int n = p_i->neighborhood[i];
        Particle *p_j = &particles[n];
        s_corr = 0.001;
        glm::vec3 b = spiky_grad(p_i->pred_p - p_j->pred_p, KERNEL_RADIUS);
        float a = (p_j->lambda + p_i->lambda + s_corr);
        delta_p += a * b;
    }
    return (1 / REST_DENSITY) * delta_p;
}

__device__ glm::vec3 vorticity(Particle *p_i, Particle *particles) {
    glm::vec3 w;
    for (int i = 0; i < p_i->num_neighbors; ++i) {
        int n = p_i->neighborhood[i];
        Particle *p_j = &particles[n];
        glm::vec3 v_ij = p_j->v - p_i->v;
        w += glm::cross(v_ij, spiky_grad(p_i->pred_p - p_j->pred_p, KERNEL_RADIUS));
    }
    return w;
}

__device__ glm::vec3 f_vorticity(Particle *p_i, Particle *particles) {
    glm::vec3 force;
    glm::vec3 w = vorticity(p_i, particles);
    float rho = rho_i(p_i, particles);
    for (int i = 0; i < p_i->num_neighbors; ++i) {
        int n = p_i->neighborhood[i];
        Particle *p_j = &particles[n];
        glm::vec3 p_x = (p_i->m * p_i->pred_p + p_j->m * p_j->pred_p) * (1.f / (p_i->m + p_j->m));
        glm::vec3 eta = p_x - p_i->pred_p;
        glm::vec3 N = glm::normalize(eta);
        force += EPS_T * glm::cross(N, w) * rho;
    }
    return force;
}

__device__ glm::vec3 XSPH_vel(Particle *p_i, Particle *particles) {
    glm::vec3 v;
    for (int i = 0; i < p_i->num_neighbors; ++i) {
        int n = p_i->neighborhood[i];
        Particle *p_j = &particles[n];
        glm::vec3 v_ij = p_j->v - p_i->v;
        v += v_ij * poly6(p_i->v - p_j->v, KERNEL_RADIUS);
    }
    return p_i->v + VISCOSITY * v;
}

__device__ void triangle_collide(Triangle &t, Particle &p) {
    glm::vec3 d = p.pred_p - p.p;
    glm::vec3 e1 = t.v2 - t.v1;
    glm::vec3 e2 = t.v3 - t.v1;
    glm::vec3 s = p.p - t.v1;
    glm::vec3 s1 = glm::cross(d, e2);
    glm::vec3 s2 = glm::cross(s, e1);
    float coeff = 1 / glm::dot(s1, e1);

    // Check if on opposite sides of plane
    float pos_dot = glm::dot(p.p - t.v1, t.n);
    float pred_dot = glm::dot(p.pred_p - t.v1, t.n);
    if (pos_dot >= 0 && pred_dot >= 0 || pos_dot < 0 && pred_dot < 0) {
        return;
    }

    // Check if inside triangle
    float b1 = coeff * glm::dot(s1, s);
    float b2 = coeff * glm::dot(s2, d);
    if (b1 >= 0 && b2 >= 0 && (1 - b1 - b2) >= 0) {
        glm::vec3 tangent = b1 * t.v2 + b2 * t.v3 + (1 - b1 - b2) * t.v1;
        if (pred_dot < 0) {
            tangent += t.n * SURFACE_OFFSET;
        } else {
            tangent -= t.n * SURFACE_OFFSET;
        }
        p.pred_p = tangent;
        p.collided = true;
        p.v = glm::reflect(p.v, t.n) * 0.5f;
    }
}

__device__ void plane_collide(Plane &p, Particle &par) {
    float pos_dot = glm::dot(par.p - p.point, p.normal);
    float pred_dot = glm::dot(par.pred_p - p.point, p.normal);
    if ((pos_dot >= 0 && pred_dot < 0) || (pos_dot < 0 && pred_dot >= 0)) {
        glm::vec3 proj_vec = par.pred_p - p.point;
        float dist = glm::dot(proj_vec, p.normal);
        if (dist < 0) {
            dist = dist - SURFACE_OFFSET;
        } else {
            dist = dist + SURFACE_OFFSET;
        }
        glm::vec3 proj_point = par.pred_p - p.normal * dist;
        par.pred_p = proj_point;
        par.v = glm::reflect(-par.v, p.normal);
    }
}