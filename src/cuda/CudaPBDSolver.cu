#include "hip/hip_runtime.h"
#include "CudaPBDSolver.h"

__global__ void seed_position(Particle *particles, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index > n) {
        return;
    }
    Particle &p = particles[index];
    p.f = vec3(0, -9.8f, 0) * p.m;
    p.v = p.v + p.f * (1.f / p.m) * dt;
    p.pred_p = p.p + p.v * dt;
}

__global__ void run_solver(Particle *particles, int n, Triangle *triangles, int num_triangles, Plane *plane, int num_planes) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index > n) {
        return;
    }
    Particle &p = particles[index];

    for (int i = 0; i < SOLVER_ITERS; ++i) {
        __syncthreads();
        p.lambda = lambda_i(&p);
        __syncthreads();

        p.dp = delta_p(&p);
        p.pred_p += p_i.dp;
        for (int i = 0; i < num_triangles; ++i) {
            triangle_collide(triangles[i], p);
        }
        for (int i = 0; i < num_planes; ++i) {
            plane_collide(planes[i], p);
        }
    }

    if (!p.collided)
        p.v = (1.f / dt) * (p.pred_p - p.p);
    __syncthreads();

    p.f += f_vorticity(&p);
    p.v = XSPH_vel(&p);
    p.collided = false;
    p.p = p.pred_p;
    p.num_neighbors = 0;
}

__device__ inline float poly6(vec3 r_ij, float h) {
    float r = length(r_ij);
    return 0 <= r && r <= h ? 315 / (64 * M_PI * pow(h, 9)) * pow(h * h - r * r, 3) : 0;
}

__device__ inline float spiky(vec3 r_ij, float h) {
    float r = length(r_ij);
    return 0 <= r && r <= h ? 15 / (M_PI * pow(h, 6)) * pow(h - r, 3) : 0;
}

__device__ inline vec3 spiky_grad(vec3 r_ij, float h) {
    float r = length(r_ij);
    if (0 < r && r <= h) {
        vec3 d = normalize(r_ij);
        float coeff = 45 / (M_PI * pow(h, 6)) * pow(h - r, 2);
        return coeff * d;
    } else {
        return vec3();
    }
}

__device__ float rho_i(Particle *p_i, Particle *particles) {
    float rho = 0;
    for (int i = 0; i < p_i->num_neighbors; ++i) {
        int n = p_i->neighborhood[i];
        Particle *p_j = &particles[i];
        rho += p_j->m * poly6(p_i->pred_p - p_j->pred_p, this->h);
    }
    return rho;
}

__device__ float lambda_i(Particle *p_i, Particle *particles) {
    float lambda = 0;
    for (int i = 0; i < p_i->num_neighbors; ++i) {
        int n = p_i->neighborhood[i];
        Particle *p_k = &particles[n];
        vec3 grad_k_ci = grad_k_Ci(p_k, p_i, particles);
        lambda += pow(length(grad_k_ci), 2);
    }
    lambda = -C_i(p_i, particles) * (1 / (lambda + this->eps_t));
    return lambda;
}

__device__ float C_i(Particle *p_i, Particle *particles) {
    return rho_i(p_i, particles) / this->rest_density - 1;
}

__device__ vec3 grad_k_Ci(Particle *p_k, Particle *p_i, Particle *particles) {
    if (p_k == p_i) {
        vec3 grad_spiky;
        for (int i = 0; i < p_i->num_neighbors; ++i) {
            int n = p_i->neighborhood[i];
            Particle *p_j = &particles[n];
            grad_spiky += spiky_grad(p_i->pred_p - p_j->pred_p, this->h);
        }
        return (1 / this->rest_density) * grad_spiky;
    } else {
        return -(1 / this->rest_density) * spiky_grad(p_i->pred_p - p_k->pred_p, this->h);
    }
}

__device__ vec3 delta_p(Particle *p_i, Particle *particles) {
    vec3 delta_p;
    float s_corr;
    for (int i = 0; i < p_i->num_neighbors; ++i) {
        int n = p_i->neighborhood[i];
        Particle *p_j = &particles[n];
        s_corr = clamp(-k * pow(poly6(p_i->pred_p - p_j->pred_p, this->h) / poly6(dq, this->h), this->n), 0.0f, 0.0001f);
        vec3 b = spiky_grad(p_i->pred_p - p_j->pred_p, this->h);
        float a = (p_j->lambda + p_i->lambda + s_corr);
        delta_p += a * b;
    }
    return (1 / this->rest_density) * delta_p;
}

__device__ vec3 vorticity(Particle *p_i, Particle *particles) {
    vec3 w;
    for (int i = 0; i < p_i->num_neighbors; ++i) {
        int n = p_i->neighbors[i];
        Particle *p_j = particles[n];
        vec3 v_ij = p_j->v - p_i->v;
        w += cross(v_ij, spiky_grad(p_i->pred_p - p_j->pred_p, this->h));
    }
    return w;
}

__device__ vec3 f_vorticity(Particle *p_i, Particle *particles) {
    vec3 force;
    vec3 w = this->vorticity(p_i, particles);
    float rho = this->rho_i(p_i, particles);
    for (int i = 0; i < p_i->num_neighbors; ++i) {
        int n = p_i->neighborhood[i];
        Particle *p_j = particles[n];
        vec3 p_x = (p_i->m * p_i->pred_p + p_j->m * p_j->pred_p) * (1.f / (p_i->m + p_j->m));
        vec3 eta = p_x - p_i->pred_p;
        vec3 N = normalize(eta);
        force += this->eps_t * cross(N, w) * rho;
    }
    return force;
}

__device__ vec3 XSPH_vel(Particle *p_i, Particle *particles) {
    vec3 v;
    for (int i = 0; i < p_i->num_neighbors; ++i) {
        int n = p_i->neighborhood[i];
        Particle *p_j = particles[n];
        vec3 v_ij = p_j->v - p_i->v;
        v += v_ij * poly6(p_i->v - p_j->v, this->h);
    }
    return p_i->v + this->c * v;
}

__device__ void triangle_collide(Triangle &t, Particle &p) {
    vec3 d = p.pred_p - p.p;
    vec3 e1 = t.v2 - t.v1;
    vec3 e2 = t.v3 - t.v1;
    vec3 s = p.p - t.v1;
    vec3 s1 = cross(d, e2);
    vec3 s2 = cross(s, e1);
    float coeff = 1 / dot(s1, e1);

    // Check if on opposite sides of plane
    float pos_dot = glm::dot(p.p - t.v1, t.n);
    float pred_dot = glm::dot(p.pred_p - t.v1, t.n);
    if (std::signbit(pos_dot) == std::signbit(pred_dot)) {
        return;
    }

    // Check if inside triangle
    float b1 = coeff * dot(s1, s);
    float b2 = coeff * dot(s2, d);
    if (b1 >= 0 && b2 >= 0 && (1 - b1 - b2) >= 0) {
        vec3 tangent = b1 * t.v2 + b2 * t.v3 + (1 - b1 - b2) * t.v1;
        if (std::signbit(pred_dot) > 0) {
            tangent += t.n * SURFACE_OFFSET;
        } else {
            tangent -= t.n * SURFACE_OFFSET;
        }
        p.pred_p = tangent;
        p.collided = true;
        p.v = glm::reflect(p.v, t.n) * 0.5f;
    }
}

__device__ void plane_collide(Plane &p, Particle &par) {
    float pos_dot = glm::dot(p.p - this->point, this->normal);
    float pred_dot = glm::dot(p.pred_p - this->point, this->normal);
    if (std::signbit(pos_dot) != std::signbit(pred_dot)) {
        glm::vec3 proj_vec = p.pred_p - this->point;
        float dist = glm::dot(proj_vec, this->normal);
        if (dist < 0) {
            dist = dist - SURFACE_OFFSET;
        } else {
            dist = dist + SURFACE_OFFSET;
        }
        glm::vec3 proj_point = p.pred_p - this->normal * dist;
        p.pred_p = proj_point;
        p.v = glm::reflect(-p.v, normal);
    }
}