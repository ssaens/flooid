#include "hip/hip_runtime.h"


#define DELTA_T 0.004f
#define PI 3.14159265359f
#define SOLVER_ITERS 3
#define KERNEL_RADIUS 0.106f
#define EPS_T 600.f
#define REST_DENSITY 6378.f
#include "CudaPBDSolver.cuh"

const int PRESSURE_POW = 4;
const float VISCOSITY = 0.01;
const float SURFACE_OFFSET = 0.000001;
const float PRESSURE_STRENGTH =  0.0005f;
const float PRESSURE_EPS = 1e-2f;
const float MAX_VORT_ADJ = 0.0001f;

using namespace glm;

__device__ float poly6(glm::vec3 r_ij, float h);
__device__ float spiky(glm::vec3 r_ij, float h);
__device__ glm::vec3 spiky_grad(glm::vec3 r_ij, float h);

__device__ float rho_i(Particle *p_i, Particle *particles);
__device__ float lambda_i(Particle *p_i, Particle *particles);
__device__ glm::vec3 grad_k_Ci(Particle *p_k, Particle *p_i, Particle *particles);
__device__ glm::vec3 delta_p(Particle *p_i, Particle *particles);
__device__ glm::vec3 vorticity(Particle *p_i, Particle *particles);
__device__ glm::vec3 vort_and_XSPH_vel(Particle *p_i, Particle *particles);

__device__ void triangle_collide(Triangle &t, Particle &p);
__device__ void plane_collide(Plane &p, Particle &par);
__device__ void sphere_collide(Particle &p);

__global__ void seed_position(Particle *particles, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= n) {
        return;
    }
    Particle &p = particles[index];
    p.v = p.v + glm::vec3(0, -9.8f, 0) * DELTA_T;
    p.pred_p = p.p + p.v * DELTA_T;
}

__global__ void run_solver(Particle *particles, int n, Triangle *triangles, int num_triangles, Plane *planes, int num_planes) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= n) {
        return;
    }
    Particle &p = particles[index];

    for (int i = 0; i < SOLVER_ITERS; ++i) {
        p.lambda = lambda_i(&p, particles);
        __syncthreads();
        p.dp = delta_p(&p, particles);
        // for (int i = 0; i < num_triangles; ++i) {
        //     triangle_collide(triangles[i], p);
        // }
       
        // sphere_collide(p);
        glm::vec3 displacement = p.pred_p + p.dp - p.p;
        if (glm::length(displacement) > 0.1f) {
            displacement = glm::normalize(displacement) * 0.1f;
        }
        p.pred_p = p.p + displacement;
         for (int i = 0; i < num_planes; ++i) {
            plane_collide(planes[i], p);
        }
            sphere_collide(p);
        __syncthreads();
    }

    // for (int i = 0; i < num_triangles; ++i) {
    //     triangle_collide(triangles[i], p);
    // }
    for (int i = 0; i < num_planes; ++i) {
        plane_collide(planes[i], p);
    }

    p.v = (1.f / DELTA_T) * (p.pred_p - p.p);
    __syncthreads();
    p.w = vorticity(&p, particles);
    __syncthreads();

    p.v = vort_and_XSPH_vel(&p, particles);
    if (glm::length(p.v) > 7.f) {
        p.v = glm::normalize(p.v) * 7.f;
    }

    p.p = p.pred_p;
    p.num_neighbors = 0;
}

__device__ float poly6(glm::vec3 r_ij, float h) {
    float r = length(r_ij);
    float out_term = h * h - r * r;
    if (out_term < MAX_VORT_ADJ) {
        return 0.f;
    }
    return 315.f / (64.f * PI * pow(h, 9)) * pow(out_term, 3);
}

__device__ glm::vec3 spiky_grad(glm::vec3 r_ij, float h) {
    float r = length(r_ij);
    float r_2 = r * r;
    if ((r_2 >= h * h) || (r_2 <= MAX_VORT_ADJ)) {
        return glm::vec3();
    }
    glm::vec3 d = normalize(r_ij);
    float coeff = 45.f / (PI * pow(h, 6)) * pow(h - r, 2);
    return coeff * d;
}

__device__ float rho_i(Particle *p_i, Particle *particles) {
    float rho = 0;
    for (int i = 0; i < p_i->num_neighbors; ++i) {
        int n = p_i->neighborhood[i];
        Particle *p_j = &particles[n];
        rho += p_j->m * poly6(p_i->pred_p - p_j->pred_p, KERNEL_RADIUS);
    }
    p_i->rho = rho;
    return rho;
}

__device__ float lambda_i(Particle *p_i, Particle *particles) {
    float C_i = rho_i(p_i, particles) / REST_DENSITY - 1;
    float grad_sum = 0;
    glm::vec3 grad_k_ci;

    for (int i = 0; i < p_i->num_neighbors; ++i) {
        int n = p_i->neighborhood[i];
        Particle *p_k = &particles[n];
        grad_k_ci = grad_k_Ci(p_k, p_i, particles);
        grad_sum += pow(glm::length(grad_k_ci), 2);
    }
    return -C_i * (1 / (grad_sum + EPS_T));
}

__device__ glm::vec3 grad_k_Ci(Particle *p_k, Particle *p_i, Particle *particles) {
    if (p_k == p_i) {
        glm::vec3 grad_spiky;
        for (int i = 0; i < p_i->num_neighbors; ++i) {
            int n = p_i->neighborhood[i];
            Particle *p_j = &particles[n];
            grad_spiky += spiky_grad(p_i->pred_p - p_j->pred_p, KERNEL_RADIUS);
        }
        return (1 / REST_DENSITY) * grad_spiky;
    } else {
        return -(1 / REST_DENSITY) * spiky_grad(p_i->pred_p - p_k->pred_p, KERNEL_RADIUS);
    }
}

__device__ glm::vec3 delta_p(Particle *p_i, Particle *particles) {
    glm::vec3 delta_p;
    float s_corr = 0.f;
    for (int i = 0; i < p_i->num_neighbors; ++i) {
        int n = p_i->neighborhood[i];
        Particle *p_j = &particles[n];
        glm::vec3 dq(0.03, 0.03, 0.03);
        float inv_sqrt_3 = 0.57735f;
        double base = poly6(p_i->pred_p - p_j->pred_p, KERNEL_RADIUS) / poly6(inv_sqrt_3 * dq, KERNEL_RADIUS);
        s_corr = -PRESSURE_STRENGTH * pow(base, PRESSURE_POW);
        glm::vec3 b = spiky_grad(p_i->pred_p - p_j->pred_p, KERNEL_RADIUS);
        float a = (p_j->lambda + p_i->lambda + s_corr);
        delta_p += a * b;
    }
    delta_p = (1 / REST_DENSITY) * delta_p;
    if (glm::length(delta_p) > 0.1) {
        delta_p = glm::normalize(delta_p) * 0.1f;
    }
    return delta_p;
}

__device__ glm::vec3 vorticity(Particle *p_i, Particle *particles) {
    glm::vec3 w;
    for (int i = 0; i < p_i->num_neighbors; ++i) {
        int n = p_i->neighborhood[i];
        Particle *p_j = &particles[n];
        glm::vec3 v_ij = p_j->v - p_i->v;
        w += glm::cross(spiky_grad(p_i->pred_p - p_j->pred_p, KERNEL_RADIUS), v_ij);
    }
    return w;
}

__device__ glm::vec3 vort_and_XSPH_vel(Particle *p_i, Particle *particles) {
    glm::vec3 eta;
    glm::vec3 neighbor_vs;
    for (int i = 0; i < p_i->num_neighbors; ++i) {
        int n = p_i->neighborhood[i];
        Particle *p_j = &particles[n];
        eta += (1 / max(p_j->rho, 100.f)) * glm::length(p_j->w) * spiky_grad(p_i->pred_p - p_j->pred_p, KERNEL_RADIUS);
        glm::vec3 v_ji = p_j->v - p_i->v;
        neighbor_vs += (1 / max(p_j->rho, 100.f)) * v_ji * poly6(p_i->pred_p - p_j->pred_p, KERNEL_RADIUS); // MAY need to flip v
    }
    if (length(eta) > MAX_VORT_ADJ) {
        eta = glm::normalize(eta);
    } else {
        eta = glm::vec3();
    }
    glm::vec3 f_vorticity = PRESSURE_EPS * glm::cross(eta, p_i->w);
    return p_i->v + VISCOSITY * neighbor_vs + DELTA_T * f_vorticity;
    // return p_i->v + VISCOSITY * neighbor_vs;
}

__device__ void triangle_collide(Triangle &t, Particle &p) {
    glm::vec3 d = p.pred_p - p.p;
    glm::vec3 e1 = t.v2 - t.v1;
    glm::vec3 e2 = t.v3 - t.v1;
    glm::vec3 s = p.p - t.v1;
    glm::vec3 s1 = glm::cross(d, e2);
    glm::vec3 s2 = glm::cross(s, e1);
    float coeff = 1 / glm::dot(s1, e1);

    // Check if on opposite sides of plane
    float pos_dot = glm::dot(p.p - t.v1, t.n);
    float pred_dot = glm::dot(p.pred_p - t.v1, t.n);
    if (pos_dot >= 0 && pred_dot >= 0 || pos_dot < 0 && pred_dot < 0) {
        return;
    }

    // Check if inside triangle
    float b1 = coeff * glm::dot(s1, s);
    float b2 = coeff * glm::dot(s2, d);
    if (b1 >= 0 && b2 >= 0 && (1 - b1 - b2) >= 0) {
        glm::vec3 tangent = b1 * t.v2 + b2 * t.v3 + (1 - b1 - b2) * t.v1;
        if (pred_dot < 0) {
            tangent += t.n * SURFACE_OFFSET;
        } else {
            tangent -= t.n * SURFACE_OFFSET;
        }
        p.pred_p = tangent;
        p.collided = true;
        // p.v = glm::reflect(p.v, t.n) * 0.5f;
        p.v = glm::vec3();
    }
}

__device__ void plane_collide(Plane &p, Particle &par) {
    float pos_dot = glm::dot(par.p - p.point, p.normal);
    float pred_dot = glm::dot(par.pred_p - p.point, p.normal);
    if ((pos_dot >= 0 && pred_dot < 0) || (pos_dot < 0 && pred_dot >= 0)) {
        glm::vec3 proj_vec = par.pred_p - p.point;
        float dist = glm::dot(proj_vec, p.normal);
        if (dist < 0) {
            dist = dist - SURFACE_OFFSET;
        } else {
            dist = dist + SURFACE_OFFSET;
        }
        glm::vec3 proj_point = par.pred_p - p.normal * dist;
        par.pred_p = proj_point;
        // par.v = glm::reflect(-par.v, p.normal);
        par.v = glm::vec3();          
    }
}

__device__ void sphere_collide(Particle &p) {
    if (p.p.y > 0) return;
    glm::vec3 origin;
    float radius = 1.5;
    float pos_dist = glm::length(p.p - origin);
    float pred_dist = glm::length(p.pred_p - origin);
    glm::vec3 normal = glm::normalize(p.pred_p - origin); // o to pred_p
    glm::vec3 ray = p.pred_p - p.p;
    if (pred_dist >=radius && pos_dist <= radius) {   
        glm::vec3 proj_point = normal * radius + origin - normal * SURFACE_OFFSET;
        p.pred_p = proj_point;
    }
}